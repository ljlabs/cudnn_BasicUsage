
#include <hip/hip_runtime.h>
#include <iostream>
#include "cudnn.h"
using namespace std;
int main(int argc, char const *argv[])
{
	cout << "hello cuda" << endl;
	cudnnStatus_t status;
	cudnnHandle_t handle;
	status = cudnnCreate(&handle);
	cout << "status " << status << endl;

// need to destroy all calls to cuda functions
	cudnnDestroy(handle);
	return 0;
}
