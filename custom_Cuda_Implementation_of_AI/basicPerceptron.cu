#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>
#include <sstream>
#include <hiprand.h>

using namespace std;
__device__ __managed__ float weightedinputs[25]; // used as list of input neuron
__device__ __managed__ float weights[25]; // used as list of neuron conection weigths
__device__ __managed__ int inputs[25]; // used as list of neuron conection weigths
__device__ __managed__ float output = 0; // used to return output
__device__ __managed__ int expctd = 0; // used to return output

__global__ void mulWeightsAndInputs() {

	weightedinputs[threadIdx.x] = weights[threadIdx.x] * inputs[threadIdx.x];

}

__global__ void feedbackward(){			// trains the weights
	float lr = 0.3;
	float error = (expctd - output);
	weights[threadIdx.x] = weights[threadIdx.x] + error * inputs[threadIdx.x] * lr;
}

void init_Weights(){
	for (int i = 0; i < 25; i++){
		weights[i] = rand() % 10 + 1;
	}
}

void feed_forward(){
	double tot = 0;
	for (int i = 0; i < 25; i++){
		tot = tot + weightedinputs[i];	
		//cout << tot << endl;
	}
	long sqtot = pow(tot, 2);
	output = 0.5*((tot / (1 + sqrt(sqtot))) + 1);

}
void cp_To_Dev(int data[25]){
	for (int i = 0; i < 25; i++){
		inputs[i] = data[i];
	}
}


int main()
{
	/*generate some training data*/
	int expected[5] = { 1, 0, 0, 0, 0 };	// lets learn the letter a
	int trainingData[5][25] = {
		{
			0, 0, 1, 0, 0,
			0, 1, 0, 1, 0,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 1,
			1, 0, 0, 0, 1,
		},
		{
			1, 1, 1, 1, 0,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 0,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 0
		},
		{
			0, 1, 1, 1, 0,
			1, 0, 0, 0, 1,
			1, 0, 0, 0, 0,
			1, 0, 0, 0, 1,
			0, 1, 1, 1, 0
		}, 
		{
			1, 1, 1, 1, 0,
			1, 0, 0, 0, 1,
			1, 0, 0, 0, 1,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 0
		},
		{
			1, 1, 1, 1, 1,
			1, 0, 0, 0, 0,
			1, 1, 1, 1, 0,
			1, 0, 0, 0, 0,
			1, 1, 1, 1, 1
		}
	};
	// initilise the weights
	init_Weights();

	for (int epoc = 0; epoc < 1000; epoc++){	// i want this to run 1000 time to ensure a good train
		for (int nRow = 0; nRow < 5; nRow++){

			// copy inputs to device
			cp_To_Dev(trainingData[nRow]);
			mulWeightsAndInputs << < 1, 25 >> >();
			hipDeviceSynchronize();
			feed_forward();
			// learn
			expctd = expected[nRow];
			feedbackward << < 1, 25 >> >();
			hipDeviceSynchronize();
		}
	}

	int TestData[4][25] = {
		{// broken A
			0, 0, 0, 0, 0,
			0, 1, 0, 1, 0,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 1,
			0, 1, 0, 0, 1,
		}, { // a B also broken
			1, 1, 1, 1, 0,
			1, 0, 1, 0, 1,
			1, 1, 0, 0, 1,
			1, 0, 0, 0, 1,
			1, 1, 1, 1, 0
		}, { //  a very broken A
			0, 0, 1, 0, 0,
			0, 1, 0, 1, 0,
			1, 0, 1, 0, 1,
			1, 1, 0, 0, 1,
			1, 0, 0, 0, 1,
		},
	};
	
	for (int i = 0; i < 3; i++){
		// test the network
		cp_To_Dev(TestData[i]);
		mulWeightsAndInputs << < 1, 25 >> >();
		hipDeviceSynchronize();
		feed_forward();
		cout << output << endl;
	}

	// this just forces the program to wait until i have completed looking at the ouput
	int wait;
	cin >> wait;
    return 0;
}
