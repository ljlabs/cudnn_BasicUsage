#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <C:\Users\jorda_000\Documents\cuda\libs\cudnn-6.5-win-v2\hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <sstream>

using namespace std;
__device__ __managed__ int ret[1000]; // managed varaible

__global__ void AplusB(int a, int b) {
	ret[threadIdx.x] = threadIdx.x * a+ b + threadIdx.x; 
}


int main()
{
	AplusB <<< 1, 1000 >>>(5, 10);
	hipDeviceSynchronize();
	for (int i = 0; i < 1000; i++){
		cout << ret[i] << endl;
	}
	int a;
	cin >> a;
    return 0;
}
